#include <stdio.h>
#include "h_colorToGreyscale.h"
#include "CHECK.h"

#define CHANNELS 3

/*
   h_colorToGreyscale
   Performs the greyscale of an image on the CPU.
   Pout array is filled with the greyscale of each pixel.
   Pin array contains the color pixels.
   width and height are the dimensions of the image.
*/
float h_colorToGreyscale(unsigned char * Pout, unsigned char * Pin,
                        int width, int height)
{

    int i, j, inIdx = 0, outIdx = 0;
    hipEvent_t start_cpu, stop_cpu;
    float cpuMsecTime = -1;

    //Use cuda functions to do the timing 
    //create event objects
    CHECK(hipEventCreate(&start_cpu));
    CHECK(hipEventCreate(&stop_cpu));
    //record the starting time
    CHECK(hipEventRecord(start_cpu));

    for (j = 0; j < height; ++j)
    {
        for (i = 0; i < width; i++, inIdx+=CHANNELS, outIdx++)
        {
            static unsigned char red, green, blue;
            red = Pin[inIdx];  
            green = Pin[inIdx + 1];  
            blue = Pin[inIdx + 2];  
            //one character in the output array is calculated based
            //upon three characters (one pixel) in the input array
            Pout[outIdx] = 0.21f*red + 0.71f*green + 0.07f*blue;
        }
    }

    //record the ending time and wait for event to complete
    CHECK(hipEventRecord(stop_cpu));
    CHECK(hipEventSynchronize(stop_cpu));
    //calculate the elapsed time between the two events 
    CHECK(hipEventElapsedTime(&cpuMsecTime, start_cpu, stop_cpu));
    return cpuMsecTime;
}
