#include "hip/hip_runtime.h"
#include <sys/stat.h>
#include <stdlib.h>
#include <stdio.h>
#include <jpeglib.h>
#include <jerror.h>
#include "wrappers.h"
#include "h_colorToGreyscale.h"
#include "d_colorToGreyscale.h"

#define CHANNELS 3

//prototypes for functions in this file 
void parseCommandArgs(int, char **, int *, int *, char **);
void printUsage();
void readJPGImage(char *, unsigned char **, int *, int *);
void writeJPGImage(char *, unsigned char *, int, int);
char * buildFilename(char *, const char *);
void compare(unsigned char * d_Pout, unsigned char * h_Pout, int size);

/*
    main 
    Opens the jpg file and reads the contents.  Uses the CPU
    and the GPU to perform the greyscale.  Compares the CPU and GPU
    results.  Writes the results to output files.  Outputs the
    time of each.
*/
int main(int argc, char * argv[])
{
    unsigned char * Pin;
    char * fileName;
    int width, height, blkWidth, blkHeight;
    parseCommandArgs(argc, argv, &blkWidth, &blkHeight, &fileName);
    readJPGImage(fileName, &Pin, &width, &height);

    //use the CPU to perform the greyscale
    unsigned char * h_Pout; 
    h_Pout = (unsigned char *) Malloc(sizeof(unsigned char) * width * height);
    float cpuTime = h_colorToGreyscale(h_Pout, Pin, width, height);
    char * h_outfile = buildFilename(fileName, "h_grey");
    writeJPGImage(h_outfile, h_Pout, width, height);

    //use the GPU to perform the greyscale 
    unsigned char * d_Pout; 
    d_Pout = (unsigned char *) Malloc((sizeof(unsigned char) * width * height));
    float gpuTime = d_colorToGreyscale(d_Pout, Pin, width, height, blkWidth, blkHeight);
    char * d_outfile = buildFilename(fileName, "d_grey");
    writeJPGImage(d_outfile, d_Pout, width, height);

    //compare the CPU and GPU results
    compare(d_Pout, h_Pout, width * height);

    printf("CPU time: %f msec\n", cpuTime);
    printf("GPU time: %f msec\n", gpuTime);
    printf("Speedup: %f\n", cpuTime/gpuTime);
    return EXIT_SUCCESS;
}

/* 
    compare
    This function takes two arrays of greyscale pixel values.  One array
    contains pixel values calculated  by the GPU.  The other array contains
    greyscale pixel values calculated by the CPU.  This function checks to
    see that the values are the same within a slight margin of error.

    d_Pout - pixel values calculated by GPU
    h_Pout - pixel values calculated by CPU
    size - size in elements of both arrays
    
    Outputs an error message and exits program if the arrays differ.
*/
void compare(unsigned char * d_Pout, unsigned char * h_Pout, int size)
{
    int i;
    for (i = 0; i < size; i++)
    {
        //GPU and CPU have different floating point standards so
        //the results could be slightly different
        int diff = d_Pout[i] - h_Pout[i];
        if (abs(diff) > 1)
        {
            printf("Greyscale results don't match.\n");
            printf("CPU pixel %d: %d\n", i, h_Pout[i]);
            printf("GPU pixel %d: %d\n", i, d_Pout[i]);
            exit(EXIT_FAILURE);
        }
    }
}

/* 
    writeJPGImage
    Writes a greyscale jpg image to an output file.

    outfile - name of jpg file (ends with a .jpg extension)
    Pout - array of pixels
    width - width (x-dimension) of image
    height - height (y-dimension) of image
*/
void writeJPGImage(char * filename, unsigned char * Pout, 
                   int width, int height)
{
   struct jpeg_compress_struct cinfo;
   struct jpeg_error_mgr jerr;
   JSAMPROW rowPointer[1];

   //set up error handling
   cinfo.err = jpeg_std_error(&jerr);
   //initialize the compression object
   jpeg_create_compress(&cinfo);

   //open the output file
   FILE * fp;
   if ((fp = fopen(filename, "wb")) == NULL)
   {
     fprintf(stderr, "Can't open %s\n", filename);
     exit(1);
   }
   //initalize state for output to outfile
   jpeg_stdio_dest(&cinfo, fp);

   cinfo.image_width = width;    /* image width and height, in pixels */
   cinfo.image_height = height;
   cinfo.input_components = 1;   /* # of color components per pixel */
   cinfo.in_color_space = JCS_GRAYSCALE;
   jpeg_set_defaults(&cinfo);
   jpeg_set_quality(&cinfo, 75, TRUE);

   //TRUE means it will write a complete interchange-JPEG file
   jpeg_start_compress(&cinfo, TRUE);

   while (cinfo.next_scanline < cinfo.image_height)
   {
      rowPointer[0] = &Pout[cinfo.next_scanline * width];
      (void) jpeg_write_scanlines(&cinfo, rowPointer, 1);
   }
   jpeg_finish_compress(&cinfo);
   fclose(fp);
   jpeg_destroy_compress(&cinfo);
}

/*
    buildFilename
    This function returns the concatenation of two strings by
    first allocating enough space to hold both strings and then
    copying the two strings into the allocated space.  
    It is used by the program to build the output file names.
*/    
char * buildFilename(char * infile, const char * prefix)
{
   int len = strlen(infile) + strlen(prefix) + 1;
   char * outfile = (char *) Malloc(sizeof(char *) * len);
   strncpy(outfile, prefix, strlen(prefix));
   strncpy(&outfile[strlen(prefix)], infile, strlen(infile) + 1);
   return outfile;
}
   
/*
    readJPGImage
    This function opens a jpg file and reads the contents.  
    Each pixel consists of bytes for red, green, and blue.  
    The array Pin is initialized to the pixel bytes.  width and height
    are pointers to ints that are set to those values.
    filename - name of the .jpg file
*/
void readJPGImage(char * filename, unsigned char ** Pin, 
                  int * width, int * height)
{
   unsigned long dataSize;             // length of the file
   int channels;                       //  3 =>RGB   4 =>RGBA 
   unsigned char * rowptr[1];          // pointer to an array
   unsigned char * jdata;              // data for the image
   struct jpeg_decompress_struct info; //for our jpeg info
   struct jpeg_error_mgr err;          //the error handler

   FILE * fp = fopen(filename, "rb"); //read binary
   if (fp == NULL)
   {
      fprintf(stderr, "Error reading file %s\n", filename);
      printUsage();
   }

   info.err = jpeg_std_error(& err);
   jpeg_create_decompress(&info);

   jpeg_stdio_src(&info, fp);
   jpeg_read_header(&info, TRUE);   // read jpeg file header
   jpeg_start_decompress(&info);    // decompress the file

   //set width and height
   (*width) = info.output_width;
   (*height) = info.output_height;
   channels = info.num_components;
   if (channels != CHANNELS)
   {
      fprintf(stderr, "%s is not an RGB jpeg image\n", filename);
      printUsage();
   }

   dataSize = (*width) * (*height) * channels;
   jdata = (unsigned char *)Malloc(dataSize);
   while (info.output_scanline < info.output_height) // loop
   {
      // Enable jpeg_read_scanlines() to fill our jdata array
      rowptr[0] = (unsigned char *)jdata +  // secret to method
                  channels * info.output_width * info.output_scanline;

      jpeg_read_scanlines(&info, rowptr, 1);
   }
   jpeg_finish_decompress(&info);   //finish decompressing
   jpeg_destroy_decompress(&info);
   fclose(fp);                      //close the file
   (*Pin) = jdata;
   return;
}

/*
    parseCommandArgs
    This function parses the command line arguments. The program can be executed in
    one of two ways:
    ./greyscalar <file>.jpg
    or
    ./greyscalar -w <blkWidth> -h <blkHeight> <file>.jpg
    This function parses the command line arguments, setting block width and block
    height to the command line argument values or to 16 if no command line arguments
    are provided.  In addition, it checks to see if the last command line argument
    is a jpg file and sets (*fileNm) to argv[i] where argv[i] is the name of the jpg
    file.  
*/
void parseCommandArgs(int argc, char * argv[], int * blkWidth, int * blkHeight, char ** fileNm)
{
    int fileIdx = 1, blkW = 16, blkH = 16;
    struct stat buffer;
    if (argc != 2 && argc != 6) printUsage();
    if (argc == 6) 
    {
        fileIdx = 5;
        if (strncmp("-bw", argv[1], 2) != 0) printUsage();
        if (strncmp("-bh", argv[3], 2) != 0) printUsage();
        blkW = atoi(argv[2]);
        blkH = atoi(argv[4]);
        if (blkW <= 0 || blkH <= 0) printUsage();
    }

    int len = strlen(argv[fileIdx]);
    if (len < 5) printUsage();
    if (strncmp(".jpg", &argv[fileIdx][len - 4], 4) != 0) printUsage();

    //stat function returns 1 if file does not exist
    if (stat(argv[fileIdx], &buffer)) printUsage();
    (*blkWidth) = blkW;
    (*blkHeight) = blkH;
    (*fileNm) = argv[fileIdx];
}

/*
    printUsage
    This function is called if there is an error in the command line
    arguments or if the .jpg file that is provided by the command line
    argument is improperly formatted.  It prints usage information and
    exits.
*/
void printUsage()
{
    printf("This application takes as input the name of a .jpg\n");
    printf("file containing a color image and creates a file\n");
    printf("containing a greyscale version of the file.\n");
    printf("\nusage: greyscaler [-bw <blkWidth> -bh <blkHeight>] <name>.jpg\n");
    printf("         <blkWidth> is the width of the blocks created for GPU\n");
    printf("         <blkHeight> is the height of the blocks created for GPU\n");
    printf("         If the -bw and -bh arguments are omitted, the block size\n");
    printf("         defaults to 16 by 16.\n");
    printf("         <name>.jpg is the name of the input jpg file\n");
    printf("Examples:\n");
    printf("./greyscaler color1200by800.jpg\n");
    printf("./greyscaler -bw 8 -bh 16 color1200by800.jpg\n");
    exit(EXIT_FAILURE);
}
